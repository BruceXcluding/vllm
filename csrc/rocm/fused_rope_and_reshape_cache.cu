#include "hip/hip_runtime.h"
#include <torch/all.h>
#include <ATen/cuda/HIPContext.h>
#include <c10/cuda/CUDAGuard.h>

#include "cuda_compat.h"
#include "dispatch_utils.h"

#ifdef USE_ROCM
  #include "quantization/fp8/amd/quant_utils.cuh"
#else
  #include "quantization/fp8/nvidia/quant_utils.cuh"
#endif

#include <algorithm>
#include <cassert>
#include <map>
#include <vector>

#ifdef USE_ROCM
  #include <hip/hip_bf16.h>
typedef __hip_bfloat16 __hip_bfloat16;
#endif

namespace vllm {

template <typename scalar_t, bool IS_NEOX>
inline __device__ void apply_token_rotary_embedding(
    const scalar_t* __restrict__ arr, const scalar_t* __restrict__ cos_ptr,
    const scalar_t* __restrict__ sin_ptr, int rot_offset, int embed_dim,
    int &x_index, scalar_t &x, int &y_index, scalar_t &y) {
  scalar_t cos, sin;
  if (IS_NEOX) {
    // GPT-NeoX style rotary embedding.
    x_index = rot_offset;
    y_index = embed_dim + rot_offset;
    cos = VLLM_LDG(cos_ptr + x_index);
    sin = VLLM_LDG(sin_ptr + x_index);
  } else {
    // GPT-J style rotary embedding.
    x_index = 2 * rot_offset;
    y_index = 2 * rot_offset + 1;
    cos = VLLM_LDG(cos_ptr + x_index / 2);
    sin = VLLM_LDG(sin_ptr + x_index / 2);
  }

  x = x * cos - y * sin;
  y = y * cos + x * sin;
}

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
inline __device__ void store_value_into_key_cache(
      const int head_size, const int num_kv_heads, 
      const int64_t block_idx, const int block_size, const int64_t block_offset
      const int x,
      cache_t* __restrict__ key_cache, int &idx, scalar_t &val) {
  const int head_idx = idx / head_size;
  const int head_offset = idx % head_size;
  const int x_idx = head_offset / x;
  const int x_offset = head_offset % x;

  const int64_t  tgt_key_idx = 
      block_idx * num_kv_heads * (head_size / x) * block_size * x +
      head_idx * (head_size / x) * block_size * x + x_idx * block_size * x +
      block_offset * x + x_offset;

  if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
    key_cache[tgt_key_idx] = val;
  } else {
    key_cache[tgt_key_idx] =
        fp8::scaled_convert<cache_t, scalar_t, kv_dt>(val, kv_scale);
  }
}

template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt>
inline __device__ void store_value_into_value_cache(
      const int head_size, const int num_kv_heads, 
      const int64_t block_idx, const int block_size, const int64_t block_offset
      cache_t* __restrict__ key_cache, int idx, scalar_t val) {
  const int head_idx = idx / head_size;
  const int head_offset = idx % head_size;

  const int64_t tgt_value_idx =
      block_idx * num_kv_heads * head_size * block_size +
      head_idx * head_size * block_size + head_offset * block_size +
      block_offset;

  if constexpr (kv_dt == Fp8KVCacheDataType::kAuto) {
    value_cache[tgt_value_idx] = val;
  } else {
    value_cache[tgt_value_idx] =
        fp8::scaled_convert<cache_t, scalar_t, kv_dt>(val, kv_scale);
  }


template <typename scalar_t, typename cache_t, Fp8KVCacheDataType kv_dt, bool IS_NEOX>
__global__ void fused_rotary_embedding_and_reshape_cache_kernel(
        const scalar_t* __restrict__ query,  // [batch_size, seq_len, num_heads, head_size] or 
                                             // [num_tokens, num_heads, head_size]
        const scalar_t* __restrict__ key,    // [num_tokens, num_heads, head_size]
        const scalar_t* __restrict__ value,  // [num_tokens, num_heads, head_size]
        cache_t* __restrict__ key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
        cache_t* __restrict__ value_cache,   // [num_blocks, num_heads, head_size, block_size]
        const scalar_t* __restrict__ cos_sin_cache,  // [max_position, 2, rot_dim // 2]
        const int64_t* __restrict__ positions,  // [batch_size, seq_len] or [num_tokens]
        const int64_t* __restrict__ slot_mapping,  // [num_tokens]
        const int64_t query_stride, const int key_stride, const int value_stride, 
        const int num_heads, const int num_kv_heads, const int head_size, 
        const int rot_dim, const int block_size, const int x, 
        const float k_scale, const float v_scale) {
    
    // Each thread block is responsible for one token.
  const int token_idx = blockIdx.x;
  int64_t pos = positions[token_idx];
  const scalar_t* cache_ptr = cos_sin_cache + pos * rot_dim;

  const int embed_dim = rot_dim / 2;
  const scalar_t* cos_ptr = cache_ptr;
  const scalar_t* sin_ptr = cache_ptr + embed_dim;

  const int nq = num_heads * embed_dim;
  for (int i = threadIdx.x; i < nq; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int rot_offset = i % embed_dim;
    const int64_t token_head = token_idx * query_stride + head_idx * head_size;
    apply_token_rotary_embedding_and_cache<scalar_t, cache_t, IS_NEOX>(
        query + token_head, cos_ptr, sin_ptr, rot_offset, embed_dim);
  }

  const int64_t slot_idx = slot_mapping[token_idx];
  const int64_t block_idx = slot_idx / block_size;
  const int64_t block_offset = slot_idx % block_size;

  const int nk = num_kv_heads * embed_dim;
  for (int i = threadIdx.x; i < nk; i += blockDim.x) {
    const int head_idx = i / embed_dim;
    const int rot_offset = i % embed_dim;
    const int64_t token_head = token_idx * key_stride + head_idx * head_size;

    int x_index, y_index;
    scalar_t x_value, y_value; 
    apply_token_rotary_embedding_and_cache<scalar_t, IS_NEOX>(
                                key + token_head, cos_ptr, sin_ptr, 
                                rot_offset, embed_dim, 
                                x_index, x_value, y_index, y_value);

    store_value_into_key_cache(head_size, num_kv_heads, block_idx, 
                                block_size, block_offset, 
                                key_cache, x_index, x_value);
    store_value_into_key_cache(head_size, num_kv_heads, block_idx, 
                                block_size, block_offset, 
                                key_cache, key_cache, y_index, y_value);
  }

  const int n = num_heads * head_size;
  for (int i = threadIdx.x; i < n; i += blockDim.x) {
    const int64_t src_value_idx = token_idx * value_stride + i;
    scalar_t value = value[src_value_idx];

    store_value_into_value_cache(head_size, num_kv_heads, block_idx, 
                                block_size, block_offset, x,
                                value_cache, i, value);
  }
}
} // namespace vllm


 // KV_T is the stored data type of kv-cache.
 // CACHE_T is the data type of key and value tensors.
 // KV_DTYPE is the real data type of kv-cache.
 // IS_NEOX
 #define CALL_ROTARY_EMBEDDING_RESHAPE_AND_CACHE(KV_T, CACHE_T, KV_DTYPE, IS_NEOX)          \
    vllm::fused_rotary_embedding_and_reshape_cache_kernel<KV_T, CACHE_T, KV_DTYPE, IS_NEOX> \
        <<<grid, block, 0, stream>>>(                                       \
                reinterpret_cast<KV_T*>(query.data_ptr()),                  \
                reinterpret_cast<KV_T*>(key.data_ptr()),                    \
                reinterpret_cast<KV_T*>(value.data_ptr()),                  \
                reinterpret_cast<CACHE_T*>(key_cache.data_ptr()),           \
                reinterpret_cast<CACHE_T*>(value_cache.data_ptr()),         \
                reinterpret_cast<KV_T*>(cos_sin_cache.data_ptr()),          \
                positions.data_ptr<int64_t>(),                              \
                slot_mapping.data_ptr<int64_t>(),                           \
                query_stride, key_stride, value_stride,                     \
                num_heads, num_kv_heads, head_size,                         \
                rot_dim, block_size, x, k_scale, v_scale);


  // The following macro is used to dispatch the conversion function based on
  // the data type of the key and value cache. The FN is a macro that calls a
  // function with template<typename scalar_t, typename cache_t,
  // Fp8KVCacheDataType kv_dt, IS_NEOX>.
  #define DISPATCH_ROPE_BY_KV_CACHE_DTYPE(SRC_DTYPE, KV_DTYPE, IS_NEOX, FN)           \
    if (KV_DTYPE == "auto") {                                                       \
      if (SRC_DTYPE == at::ScalarType::Float) {                                     \
        FN(float, float, vllm::Fp8KVCacheDataType::kAuto, IS_NEOX);                 \
      } else if (SRC_DTYPE == at::ScalarType::Half) {                               \
        FN(uint16_t, uint16_t, vllm::Fp8KVCacheDataType::kAuto, IS_NEOX);           \
      } else if (SRC_DTYPE == at::ScalarType::BFloat16) {                           \
        FN(__hip_bfloat16, __hip_bfloat16, vllm::Fp8KVCacheDataType::kAuto, IS_NEOX); \
      } else {                                                                      \
        TORCH_CHECK(false, "Unsupported input type of kv cache: ", SRC_DTYPE);      \
      }                                                                             \
    } else {                                                                        \
      if (KV_DTYPE == "fp8" || KV_DTYPE == "fp8_e4m3") {                            \
        if (SRC_DTYPE == at::ScalarType::Float) {                                   \
          FN(float, uint8_t, vllm::Fp8KVCacheDataType::kFp8E4M3, IS_NEOX);          \
        } else if (SRC_DTYPE == at::ScalarType::Half) {                             \
          FN(uint16_t, uint8_t, vllm::Fp8KVCacheDataType::kFp8E4M3, IS_NEOX);       \
        } else if (SRC_DTYPE == at::ScalarType::BFloat16) {                         \
          FN(__hip_bfloat16, uint8_t, vllm::Fp8KVCacheDataType::kFp8E4M3, IS_NEOX);  \
        } else {                                                                    \
          TORCH_CHECK(false,                                                        \
                      "Unsupported input type of kv cache: ", SRC_DTYPE);           \
        }                                                                           \
      } else if (KV_DTYPE == "fp8_e5m2") {                                          \
        if (SRC_DTYPE == at::ScalarType::Float) {                                   \
          FN(float, uint8_t, vllm::Fp8KVCacheDataType::kFp8E5M2, IS_NEOX);          \
        } else if (SRC_DTYPE == at::ScalarType::Half) {                             \
          FN(uint16_t, uint8_t, vllm::Fp8KVCacheDataType::kFp8E5M2, IS_NEOX);       \
        } else if (SRC_DTYPE == at::ScalarType::BFloat16) {                         \
          FN(__hip_bfloat16, uint8_t, vllm::Fp8KVCacheDataType::kFp8E5M2, IS_NEOX);  \
        } else {                                                                    \
          TORCH_CHECK(false,                                                        \
                      "Unsupported input type of kv cache: ", SRC_DTYPE);           \
        }                                                                           \
      } else {                                                                      \
        TORCH_CHECK(false, "Unsupported data type of kv cache: ", KV_DTYPE);        \
      }                                                                             \
    }


void fused_rotary_embedding_and_reshape_cache(
        torch::Tensor& query,   // [batch_size, seq_len, num_heads * head_size] or
                                // [num_tokens, num_heads * head_size]
        torch::Tensor& key,     // [batch_size, seq_len, num_kv_heads * head_size] or
                                // [num_tokens, num_kv_heads * head_size]
        torch::Tensor& value,   // [num_tokens, num_heads, head_size]
        torch::Tensor& key_cache,     // [num_blocks, num_heads, head_size/x, block_size, x]
        torch::Tensor& value_cache,   // [num_blocks, num_heads, head_size, block_size]
        const std::string& kv_cache_dtype,
        torch::Tensor& cos_sin_cache, // [max_position, rot_dim]
        torch::Tensor& positions,     // [batch_size, seq_len] or [num_tokens]
        torch::Tensor& slot_mapping,  // [num_tokens]
        const double k_scale,
        const double v_scale,
        bool is_neox) {
    int64_t num_tokens = query.numel() / query.size(-1);
    int rot_dim = cos_sin_cache.size(1);
    int head_size = key.size(2);
    int num_heads = query.size(-1) / head_size;
    int num_kv_heads = key.size(-1) / head_size;
    int64_t query_stride = query.stride(-2);
    int64_t key_stride = key.stride(-2);
    int64_t value_stride = value.stride(-2);

    int block_size = key_cache.size(3);
    int x = key_cache.size(4);

    dim3 grid(num_tokens);
    dim3 block(std::min<int64_t>(num_heads * rot_dim / 2, 512));
    const at::cuda::OptionalCUDAGuard device_guard(device_of(query));
    const hipStream_t stream = at::cuda::getCurrentCUDAStream();
    VLLM_DISPATCH_FLOATING_TYPES(query.scalar_type(), 
        "fused_rotary_embedding_and_reshape_cache_kernel", 
        [&] {
            if (is_neox) {
                DISPATCH_ROPE_BY_KV_CACHE_DTYPE(query.scalar_type(), 
                    kv_cache_dtype, true, CALL_ROTARY_EMBEDDING_RESHAPE_AND_CACHE)
            } else {
                DISPATCH_ROPE_BY_KV_CACHE_DTYPE(query.scalar_type(), 
                    kv_cache_dtype, false, CALL_ROTARY_EMBEDDING_RESHAPE_AND_CACHE)
            }
    });
}